#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "kernel.h"

#define SQR(x) (x)*(x)
#define PI 3.14159265358979323846

double* angs;
long* RelatedLocIndex,* RCadi;
int *d_jy, *d_jx;
int *d_dNE, *d_dNW, *d_dSE, *d_dSW;

__global__ void adjangle1(double* angs, long* Locx, long* Locy, long* SkelImwithout1, int nx, int SkelNum, int imgW) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * nx + x;
	if (tid < SkelNum) {
		int G1 = SkelImwithout1[Locx[tid] * imgW + Locy[tid] + 1];
		int G2 = SkelImwithout1[(Locx[tid] - 1) * imgW + Locy[tid] + 1];
		int G3 = SkelImwithout1[(Locx[tid] - 1) * imgW + Locy[tid]];
		int G4 = SkelImwithout1[(Locx[tid] - 1) * imgW + Locy[tid] - 1];
		int G5 = SkelImwithout1[Locx[tid] * imgW + Locy[tid] - 1];
		int G6 = SkelImwithout1[(Locx[tid] + 1) * imgW + Locy[tid] - 1];
		int G7 = SkelImwithout1[(Locx[tid] + 1) * imgW + Locy[tid]];
		int G8 = SkelImwithout1[(Locx[tid] + 1) * imgW + Locy[tid] + 1];
		int G9 = SkelImwithout1[(Locx[tid] + 1) * imgW + Locy[tid] + 2];
		int G10 = SkelImwithout1[Locx[tid] * imgW + Locy[tid] + 2];
		int G11 = SkelImwithout1[(Locx[tid] - 1) * imgW + Locy[tid] + 2];
		int G12 = SkelImwithout1[(Locx[tid] - 2) * imgW + Locy[tid] + 2];
		int G13 = SkelImwithout1[(Locx[tid] - 2) * imgW + Locy[tid] + 1];
		int G14 = SkelImwithout1[(Locx[tid] - 2) * imgW + Locy[tid]];
		int G15 = SkelImwithout1[(Locx[tid] - 2) * imgW + Locy[tid] - 1];
		int G16 = SkelImwithout1[(Locx[tid] - 2) * imgW + Locy[tid] - 2];
		int G17 = SkelImwithout1[(Locx[tid] - 1) * imgW + Locy[tid] - 2];
		int G18 = SkelImwithout1[Locx[tid] * imgW + Locy[tid] - 2];
		int G19 = SkelImwithout1[(Locx[tid] + 1) * imgW + Locy[tid] - 2];
		int G20 = SkelImwithout1[(Locx[tid] + 2) * imgW + Locy[tid] - 2];
		int G21 = SkelImwithout1[(Locx[tid] + 2) * imgW + Locy[tid] - 1];
		int G22 = SkelImwithout1[(Locx[tid] + 2) * imgW + Locy[tid]];
		int G23 = SkelImwithout1[(Locx[tid] + 2) * imgW + Locy[tid] + 1];
		int G24 = SkelImwithout1[(Locx[tid] + 2) * imgW + Locy[tid] + 2];

		int G[5][5] = { {G16, G15, G14, G13, G12}, {G17, G4, G3, G2, G11}, {G18, G5, 1, G1, G10}, {G19, G6, G7, G8, G9}, {G20, G21, G22, G23, G24} };

		int N = 0;
		int temp1 = 0;
		double meanx = 0;
		double meany = 0;
		int temp3 = 0;
		int cnt = 0;

#pragma unrolling
		for (int i = 1; i <= 5; i++) {
#pragma unrolling
			for (int j = 1; j <= 5; j++) {
				if (G[i - 1][j - 1] == 1) {
					N++;
					temp1 += i * j;
					meanx += i;
					meany += j;
					temp3 += i * i;
					cnt++;
				}
			}

		}

		meanx /= cnt;
		meany /= cnt;

		double temp2 = N * meanx * meany;
		double temp4 = N * meanx * meanx;

		double k = (temp1 - temp2) / (temp3 - temp4);

		double ang;

		if (isnan(k)) {
			ang = PI / 2;
		}
		else {
			ang = atan(k);
		}

		angs[tid] = ang;
	}
}

__global__ void PreR(long* RCadi, long* RelatedLocIndex, int* SkelImwithout1, int* SingleResult, int* rr, int PreProjectionNum, int imgH, int imgW, int nx, int skelNum, int* loc2Index) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * nx + x;
	if (tid < skelNum) {
		int* Loc = (int*)malloc(sizeof(int) * (PreProjectionNum + 7));
		Loc[0] = SingleResult[tid];

		int size = 1;

#pragma unrolling
		for (int k = 0; k < 1000; k++) {
			int presize = size;
			for (int K = 0; K < presize; K++) {
				int l = Loc[K];
				int neighbors[8] = { l - imgW - 1, l - imgW, l - imgW + 1 , l - 1, l + 1, l + imgW - 1, l + imgW , l + imgW + 1 };
#pragma unrolling                   
				for (int i = 0; i < 8; i++) {
					if (SkelImwithout1[neighbors[i]] == 1){
						bool jud = false;
						for (int j = 0; j < size; j++) {
							if (Loc[j] == neighbors[i]) {
								jud = true;
								break;
							}
						}			

						if (!jud) {
							Loc[size] = neighbors[i];
							size++;
						}
					}
				}
			}

			if (size >= PreProjectionNum) {
				for (int j = 0; j < PreProjectionNum; j++) {
					int ind = loc2Index[Loc[j]];
					RelatedLocIndex[tid * PreProjectionNum + j] = ind;
					RCadi[tid * PreProjectionNum + j] = rr[ind];
				}
				break;
			}
		}
		free(Loc);
	}
}

__global__ void computeDistance(int *jy, int *jx, int *dNE, int *dNW, int *dSE, int *dSW, int i, int j, int njunc) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (x < njunc) {
		dNE[x] = SQR(i - jy[x]) + SQR(j - jx[x]);
		dNW[x] = SQR(i - jy[x]) + SQR(j + 1 - jx[x]);
		dSE[x] = SQR(i + 1 - jy[x]) + SQR(j - jx[x]);
		dSW[x] = SQR(i + 1 - jy[x]) + SQR(j + 1 - jx[x]);
	}
}

void getAngs(long* locx, long* locy, long* SkelImwithout1, int SkelNum, int imgH, int imgW) {
	int dev = 0;
	hipSetDevice(dev);

	double* d_angs;
	long* d_locx, *d_locy, *d_SkelImwithout1;

	angs = (double*)malloc(sizeof(double) * SkelNum);

	hipMalloc((double**)&d_angs, sizeof(double) * SkelNum);
	hipMalloc((long**)&d_locx, sizeof(long) * SkelNum);
	hipMalloc((long**)&d_locy, sizeof(long) * SkelNum);
	hipMalloc((long**)&d_SkelImwithout1, sizeof(long) * imgH * imgW);

	hipMemcpy(d_locx, locx, sizeof(long) * SkelNum, hipMemcpyHostToDevice);//
	hipMemcpy(d_locy, locy, sizeof(long) * SkelNum, hipMemcpyHostToDevice);
	hipMemcpy(d_SkelImwithout1, SkelImwithout1, sizeof(long) * imgH * imgW, hipMemcpyHostToDevice);

	int nx = (int)ceil(sqrt(SkelNum));
	int ny = (int)ceil(sqrt(SkelNum));

	dim3 block(32, 32);//the size of a block must not bigger than 1024
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	adjangle1 << <grid, block >> > (d_angs, d_locx, d_locy, d_SkelImwithout1, nx, SkelNum, imgW);

	hipDeviceSynchronize();

	hipMemcpy(angs, d_angs, sizeof(double) * SkelNum, hipMemcpyDeviceToHost);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Got error %s at %s:%d\n", hipGetErrorString(err), \
			__FILE__, __LINE__); \
			// Possibly: exit(-1) if program cannot continue....
	}

	hipFree(d_angs);
	hipFree(d_locx);
	hipFree(d_locy);
	hipFree(d_SkelImwithout1);

	hipDeviceReset();
}

void getRCadiAndRelatedLocIndex(long* SkelImwithout1, long* SingleResult, long* rr, int PreProjectionNum, int imgH, int imgW, int skelNum, long* loc2Index) {
	int dev = 0;
	hipSetDevice(dev);

	long* d_RCadi, * d_RelatedLocIndex;
	int* d_SingleResult, *d_rr,*d_SkelImwithout1, *d_loc2Index;

	RCadi = (long*)malloc(sizeof(long) * (skelNum * PreProjectionNum));
	RelatedLocIndex = (long*)malloc(sizeof(long) * (skelNum * PreProjectionNum));
	memset(RCadi, 0, sizeof(long) * (skelNum * PreProjectionNum));
	memset(RelatedLocIndex, 0, sizeof(long) * (skelNum * PreProjectionNum));

	hipMalloc((long**)&d_RCadi, sizeof(long) * (skelNum * PreProjectionNum));
	hipMalloc((long**)&d_RelatedLocIndex, sizeof(long) * (skelNum * PreProjectionNum));
	hipMalloc((int**)&d_SkelImwithout1, sizeof(int) * imgH * imgW);
	hipMalloc((int**)&d_SingleResult, sizeof(int) * skelNum);
	hipMalloc((int**)&d_rr, sizeof(int) * skelNum);
	hipMalloc((int**)&d_loc2Index, sizeof(int) * imgH * imgW);

	hipMemcpy(d_RCadi, RCadi, sizeof(long) * (skelNum * PreProjectionNum), hipMemcpyHostToDevice);//
	hipMemcpy(d_RelatedLocIndex, RelatedLocIndex, sizeof(long) * (skelNum * PreProjectionNum), hipMemcpyHostToDevice);
	hipMemcpy(d_SkelImwithout1, SkelImwithout1, sizeof(int) * imgH * imgW, hipMemcpyHostToDevice);
	hipMemcpy(d_SingleResult, SingleResult, sizeof(int) * skelNum, hipMemcpyHostToDevice);
	hipMemcpy(d_rr, rr, sizeof(int) * skelNum, hipMemcpyHostToDevice);
	hipMemcpy(d_loc2Index, loc2Index, sizeof(int) * imgH * imgW, hipMemcpyHostToDevice);

	size_t neededHeapSize = (PreProjectionNum + 7) * skelNum * sizeof(int);

	hipDeviceSetLimit(hipLimitMallocHeapSize, (neededHeapSize / (1024 * 1024) + 1) * 1024 * 1024);

	int nx = (int)ceil(sqrt(skelNum));
	int ny = (int)ceil(sqrt(skelNum));

	dim3 block(32, 32);//the size of a block must not bigger than 1024
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	PreR << <grid, block >> > (d_RCadi, d_RelatedLocIndex, d_SkelImwithout1, d_SingleResult, d_rr, PreProjectionNum, imgH, imgW, nx, skelNum, d_loc2Index);

	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Got error %s at %s:%d\n", hipGetErrorString(err), \
			__FILE__, __LINE__); \
			// Possibly: exit(-1) if program cannot continue....
	}

	hipMemcpy(RCadi, d_RCadi, sizeof(long) * (skelNum * PreProjectionNum), hipMemcpyDeviceToHost);
	hipMemcpy(RelatedLocIndex, d_RelatedLocIndex, sizeof(long) * (skelNum * PreProjectionNum), hipMemcpyDeviceToHost);

	hipFree(d_RCadi);
	hipFree(d_RelatedLocIndex);
	hipFree(d_SkelImwithout1);
	hipFree(d_SingleResult);
	hipFree(d_rr);

	hipDeviceReset();
}

void init_deviceMem(int *jy, int *jx, int njunc) {
	hipMalloc((int**)&d_jy, njunc * sizeof(int));
	hipMalloc((int**)&d_jx, njunc * sizeof(int));
	hipMalloc((int**)&d_dNE, njunc * sizeof(int));
	hipMalloc((int**)&d_dNW, njunc * sizeof(int));
	hipMalloc((int**)&d_dSE, njunc * sizeof(int));
	hipMalloc((int**)&d_dSW, njunc * sizeof(int));

	hipMemcpy(d_jy, jy, njunc * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_jx, jx, njunc * sizeof(int), hipMemcpyHostToDevice);
}

void free_jy_jx() {
	hipFree(d_jy);
	hipFree(d_jx);
	hipFree(d_dNE);
	hipFree(d_dNW);
	hipFree(d_dSE);
	hipFree(d_dSW);
}

void getDistance(int i, int j, int *dNE, int *dNW, int *dSE, int *dSW, int njunc) {
	dim3 block(256);
	dim3 grid((njunc + block.x - 1) / block.x);

	computeDistance << <grid, block >> > (d_jy, d_jx, d_dNE, d_dNW, d_dSE, d_dSW, i, j, njunc);

	hipDeviceSynchronize();

	hipMemcpy(dNE, d_dNE, njunc * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(dNW, d_dNW, njunc * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(dSE, d_dSE, njunc * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(dSW, d_dSW, njunc * sizeof(int), hipMemcpyDeviceToHost);
}
